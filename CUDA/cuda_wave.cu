/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

void check_param(void);
void printfinal(void);

/**********************************************************************
 *  Checks input values from parameters
 *********************************************************************/
void check_param(int nsteps, int tpoints)
{
    char tchar[20];

    /* check number of points, number of iterations */
    while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
        printf("Enter number of points along vibrating string [%d-%d]: "
            , MINPOINTS, MAXPOINTS);
        scanf("%s", tchar);
        tpoints = atoi(tchar);
        if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
            printf("Invalid. Please enter value between %d and %d\n",
                MINPOINTS, MAXPOINTS);
    }
    while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
        printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
        scanf("%s", tchar);
        nsteps = atoi(tchar);
        if ((nsteps < 1) || (nsteps > MAXSTEPS))
            printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
    }
    printf("Using points = %d, steps = %d\n", tpoints, nsteps);
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal(float* values, int tpoints)
{
    int i;

    for (i = 1; i <= tpoints; i++) {
        printf("%6.4f ", values[i]);
        if (i % 10 == 0)
            printf("\n");
    }
}

__global__ void init_line_kernel(float* oldval, float* values, int tpoints) {
    //int i, j;
    float x, fac, k, tmp;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x; // Total number of threads
    fac = 2.0 * PI;
    tmp = (float)(tpoints - 1);

    for (int idx = tid; idx <= tpoints; idx += offset) {
        if (idx >= 1) {
            k = (float)(idx - 1);
            x = k / tmp;
            values[idx] = __sinf(fac * x);
            oldval[idx] = values[idx];
        }
    }
}

__global__ void update_kernel(float* oldval, float* values, float* newval, int nsteps, int tpoints)
{
    int i, j, tid, offset;
    tid = blockIdx.x * blockDim.x + threadIdx.x; // threadId
    offset = blockDim.x * gridDim.x; // Total number of threads

    /* Update values for each time step */
    for (i = 1; i <= nsteps; i++) {
        for (j = tid; j <= tpoints; j += offset) {
            /* global endpoints */
            if ((j == 1) || (j == tpoints)) {
                newval[j] = 0.0;
            }
            else {
                float dtime, c, dx, tau, sqtau;
                dtime = 0.3;
                c = 1.0;
                dx = 1.0;
                tau = (c * dtime / dx);
                sqtau = tau * tau;
                newval[j] = (2.0 * values[j]) - oldval[j] + (sqtau *  (-2.0)*values[j]);
            }
            /* Update old values with new values */
            oldval[j] = values[j];
            values[j] = newval[j];
        }
    }
}


/**********************************************************************
 *  Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
	int nsteps,                     /* number of time steps */
		tpoints;                    /* total points along string */
  
    /* Read Parameters */
    sscanf(argv[1], "%d", &tpoints);
    sscanf(argv[2], "%d", &nsteps);
    check_param(nsteps, tpoints);

    /* Setup Kernel Launch Parameters */
    int threadsPerBlock = 512;
    int numBlocks = (tpoints / threadsPerBlock) + 1;

    /* Allocate Host Memory */
    float *oldval, *values, *newval;
    oldval = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time (t-dt) */
    values = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time t */
    newval = (float*)malloc((tpoints + 2) * sizeof(float)); /* values at time (t+dt) */

    /* Allocate Device Memory */
    float *gpu_oldval, *gpu_values, *gpu_newval;
    hipMalloc(&gpu_oldval, (tpoints + 2) * sizeof(float));
    hipMalloc(&gpu_values, (tpoints + 2) * sizeof(float));
    hipMalloc(&gpu_newval, (tpoints + 2) * sizeof(float));

    /* Initialize Wave */
    printf("Initializing points on the line...\n");
    init_line_kernel <<<numBlocks, threadsPerBlock>>> (gpu_oldval, gpu_values, tpoints);
    
    /* Update Wave */
    printf("Updating all points for all time steps...\n");
    update_kernel <<<numBlocks, threadsPerBlock>>> (gpu_oldval, gpu_values, gpu_newval, nsteps, tpoints);

    /* Copy Memory from Device to Host */
    hipMemcpy(oldval, gpu_oldval, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(values, gpu_values, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(newval, gpu_newval, (tpoints + 2) * sizeof(float), hipMemcpyDeviceToHost);

    /* Print Result */
    printf("Printing final results...\n");
    printfinal(values, tpoints);
    printf("\nDone.\n\n");
    return 0;
}